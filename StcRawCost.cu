#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "StereoMatcher.h"
#include "Warp1D.h"

#include <hip/hip_runtime.h>

#if defined(WIN32) ||  CUDAVER >= 5
#include "hip/hip_runtime.h"
#include ""
#endif

#include "StcRawCost.h"

// Serial Execution Versions

__device__ __host__ void InterpolateLine(int buf[], int s, int w, int nB, EStereoInterpFn match_interp)     // interpolation function
{
    // Interpolate the missing values
    float si = 1.0f / s;
    for (int x = 0; x < w - 1; x++)
    {
        for (int b = 0; b < nB; b++)
        {
            int *v = &buf[s*x*nB + b];
            float I0 = v[0];
            float I1 = v[s*nB];
            if (match_interp == eCubic) // cubic interpolation
            {
                float Im = (x > 0) ? v[-s*nB] :
                    (I0 - (I1 - I0));  // extend linearly
                float Ip = (x + 1 < w - 1) ? v[2 * s*nB] :
                    (I1 + (I1 - I0));  // extend linearly
                float sf = si;
                for (int is = 1; is < s; is++, sf += si)
                {
                    v += nB;
                    float Ii = CubicInterpolate(sf, Im, I0, I1, Ip);
                    v[0] = int(Ii);
                }
            }
            else  // linear interpolation
            {
                float d = (I1 - I0) / (float)s;
                for (int is = 1; is < s; is++)
                {
                    v += nB;
                    I0 += d;
                    v[0] = int(I0);
                }
            }
        }
    }
}

__device__ __host__ void BirchfieldTomasiMinMax(const int* buffer, int* min_buf, int* max_buf, const int w, const int b)
{
    // Compute for every (interpolated) pixel, the minimum and maximum
    //  values in the two half-intervals before and after it
    //  (see [Birchfield & Tomasi, PAMI 20(40), April 1998, p. 401]).

    // Process each band separaterly
    for (int k = 0; k < b; k++)
    {
        int Ir = buffer[k], b1 = buffer[k];
        for (int x = 0, l = k; x < w; x++, l += b)
        {
            int Il = Ir, b0 = b1;   // shift down previously computed values
            if (x < w - 1)
                b1 = buffer[l + b];
            Ir = (b0 + b1 + 1) / 2;   // interpolated half-value
            min_buf[l] = __min(Il, __min(b0, Ir));
            max_buf[l] = __max(Il, __max(b0, Ir));
        }
    }
}

__device__ __host__ void MatchLine(int w, int b, int interpolated,
    int rmn[], int rmx[],     // min/max of ref (ref if rmx == 0)
    int mmn[], int mmx[],     // min/max of mtc (mtc if mmx == 0)
    float cost[],
    int m_disp_n, int disp, int disp_den,
    EStereoMatchFn match_fn,  // matching function
    int match_max,            // maximum difference for truncated SAD/SSD
    float match_outside)        // special value for outside match
{
    // Set up the starting addresses, pointers, and cutoff value
    int n = (w - 1)*disp_den + 1;             // number of reference pixels
    int s = (interpolated) ? 1 : disp_den;     // skip in reference pixels
    std::vector<float> cost1;
    cost1.resize(n);
    int cutoff = (match_fn == eSD) ? match_max * match_max : abs(match_max);
    // TODO:  cutoff is not adjusted for the number of bands...
    const float bad_cost = -1;

    // Match valid pixels
    float  left_cost = bad_cost;
    float right_cost = bad_cost;
    int x, y;
    for (x = 0; x < n; x += s)
    {
        // Compute ref and match pointers
        cost1[x] = bad_cost;
        int x_r = x, x_m = x + disp;
        if (x_m < 0 || x_m >= n)
            continue;
        int* rn = &rmn[x_r*b];    // pointer to ref or min pixel(s)
        int* rx = &rmx[x_r*b];    // pointer to ref    max pixel(s)
        int* mn = &mmn[x_m*b];    // pointer to mtc or min pixel(s)
        int* mx = &mmx[x_m*b];    // pointer to mtc    max pixel(s)
        int  diff_sum = 0;        // accumulated error

        // This code could be special-cased for b==1 for more efficiency...
        for (int ib = 0; ib < b; ib++)
        {
            int diff1 = mn[ib] - rn[ib];    // straightforward difference
            if (rmx && mmx)
            {
                // Compare intervals (see partial shuffle code in StcEvaluate.cpp)
                int xn = __max(rn[ib], mn[ib]);     // max of mins
                int nx = __min(rx[ib], mx[ib]);     // min of maxs
                if (xn <= nx)
                    diff1 = 0;          // overlapping ranges -> no error
                else
                    diff1 = (mn[ib] > rx[ib]) ?     // check sign
                    mn[ib] - rx[ib] :
                    rn[ib] - mx[ib];          // gap between intervals
            }
            int diff2 = (match_fn == eSD) ?    // squared or absolute difference
                diff1 * diff1 : abs(diff1);
            diff_sum += diff2;
        }
        int diff3 = __min(diff_sum, cutoff);    // truncated difference
        if (left_cost == bad_cost)
            left_cost = diff3;  // first cost computed
        right_cost = diff3;     // last  cost computed
        cost1[x] = diff3;        // store in temporary array
    }

    // Fill in the left and right edges
    if (UNDEFINED_COST)
        left_cost = right_cost = match_outside;
    for (x = 0; x < n && cost1[x] == bad_cost; x += s)
        cost1[x] = left_cost;
    for (x = n - 1; x >= 0 && cost1[x] == bad_cost; x -= s)
        cost1[x] = right_cost;

    // Box filter if interpolated costs
    int dh = disp_den / 2;
    float box_scale = 1.0 / (2 * dh + 1);
    for (x = 0, y = 0; y < w*m_disp_n; x += disp_den, y += m_disp_n)
    {
        if (interpolated && disp_den > 1)
        {
            float sum = 0;
            for (int k = -dh; k <= dh; k++)
            {
                int l = __max(0, __min(n - 1, x + k));  // TODO: make more efficient
                sum += cost1[l];
            }
            cost[y] = int(box_scale * sum + 0.5);
        }
        else
            cost[y] = cost1[x];
    }
}

// Parallel Execution Versions

__global__ void BirchfieldTomasiMinMaxKernel(const int* buffer, int* min_buf, int* max_buf, const int w, const int b, int buffer_length)
{
    // Compute for every (interpolated) pixel, the minimum and maximum
    //  values in the two half-intervals before and after it
    //  (see [Birchfield & Tomasi, PAMI 20(40), April 1998, p. 401]).

    __shared__ int buffer_s[BLOCKSIZE];

    unsigned k = threadIdx.x + blockIdx.x * blockDim.x;
    //unsigned x = threadIdx.y + blockIdx.y * blockDim.y;

    if (k < b)
    {
        for (int x = 0, l = k; x < w; x++, l += b)
        {
            buffer_s[l] = buffer[l];
        }
    }
    __syncthreads();

    // Process each band separately
    if (k < b)
    {
        int Ir = buffer[k], b1 = buffer[k];
        for (int x = 0, l = k; x < w; x++, l += b)
        {
            int Il = Ir, b0 = b1;   // shift down previously computed values
            if (x < w - 1)
                b1 = buffer[l + b];
            Ir = (b0 + b1 + 1) / 2;   // interpolated half-value
            min_buf[l] = __min(Il, __min(b0, Ir));
            max_buf[l] = __max(Il, __max(b0, Ir));
        }
    }
}

void BirchfieldTomasiMinMax(const int* buffer, int* min_buf_d, int* max_buf_d, const int w, const int b, int buffer_length)
{
    dim3 gridSize, blockSize(BLOCKSIZE, 1, 1);
    gridSize.x = (unsigned int)ceil((float)(w*b) / (float)blockSize.x);

    int* buffer_d;

    hipMalloc(&min_buf_d, buffer_length*sizeof(int));
    hipMalloc(&max_buf_d, buffer_length*sizeof(int));
    hipMalloc(&buffer_d, buffer_length*sizeof(int));

    hipMemcpy(buffer_d, buffer, w*b*sizeof(int), hipMemcpyHostToDevice);

    BirchfieldTomasiMinMaxKernel<<<gridSize, blockSize>>>(buffer_d, min_buf_d, max_buf_d, w, b, buffer_length);
    hipDeviceSynchronize();

    // Don't copy anything back since next call to MatchLine is parallelized
}


void MatchLine(int w, int b, int interpolated,
    int* rmn, int* rmx,     // min/max of ref (ref if rmx == 0)
    int* mmn, int* mmx,     // min/max of mtc (mtc if mmx == 0)
    float* cost,
    int m_disp_n, int disp, int disp_den,
    EStereoMatchFn match_fn,  // matching function
    int match_max,            // maximum difference for truncated SAD/SSD
    float match_outside,        // special value for outside match
    int match_interval,
    int match_interpolated,
    int buffer_length) // length of the rmn, rmx, mmn, mmx buffers
{
    // Set up the starting addresses, pointers, and cutoff value
    int n = (w - 1)*disp_den + 1;             // number of reference pixels
    int s = (interpolated) ? 1 : disp_den;     // skip in reference pixels
    float* cost1 = (float*)malloc(n*sizeof(float));
    int cutoff = (match_fn == eSD) ? match_max * match_max : abs(match_max);
    // TODO:  cutoff is not adjusted for the number of bands...
    const float bad_cost = -1;

    int cost1_length = n;
    int cost_length = w * m_disp_n;

    // Match valid pixels
    float  left_cost = bad_cost;
    float right_cost = bad_cost;

    MatchPixels(w, b, interpolated, rmn, rmx, mmn, mmx, cost1, disp, match_fn, n, s, cutoff, match_interval, match_interpolated, buffer_length, cost1_length);

    // left & right cost search
    for (int x = 0; x < n; x++)
    {
        if (cost1[x] != bad_cost)
        {
            left_cost = cost1[x];
            break;
        }
    }
    for (int x = n - 1; x >= 0; x--)
    {
        if (cost1[x] != bad_cost)
        {
            right_cost = cost1[x];
            break;
        }
    }

    // Fill in the left and right edges
    if (UNDEFINED_COST)
        left_cost = right_cost = match_outside;

    for (int x = 0; x < cost1_length && cost1[x] == bad_cost; x += s)
        cost1[x] = left_cost;
    for (int x = cost1_length - 1; x >= 0 && cost1[x] == bad_cost; x -= s)
        cost1[x] = right_cost;

    // Box filter if interpolated costs
    BoxFilter(cost1, cost, n, w, m_disp_n, disp_den, interpolated, cost1_length, cost_length);
}


__global__ void MatchPixelsKernel(int w, int b, int interpolated,
    int* rmn, int* rmx,     // min/max of ref (ref if rmx == 0)
    int* mmn, int* mmx,     // min/max of mtc (mtc if mmx == 0)
    float* cost1,
    int disp,
    EStereoMatchFn match_fn,  // matching function
    int n,
    int s,
    int cutoff,
    int buffer_length,
    int cost1_length)
{

    unsigned x = (threadIdx.x + blockIdx.x * blockDim.x) * s;

    // Match valid pixels
    if (x < n)
    {
        // Compute ref and match pointers
        cost1[x] = BAD_COST;
        int x_r = x;
        int x_m = x + disp;
        if (x_m >= 0 && x_m < n)
        {
            int* rn = &rmn[x_r*b];    // pointer to ref or min pixel(s)
            int* rx = &rmx[x_r*b];    // pointer to ref    max pixel(s)
            int* mn = &mmn[x_m*b];    // pointer to mtc or min pixel(s)
            int* mx = &mmx[x_m*b];    // pointer to mtc    max pixel(s)
            int  diff_sum = 0;        // accumulated error

            // This code could be special-cased for b==1 for more efficiency...
            for (int ib = 0; ib < b; ib++)
            {
                int diff1 = mn[ib] - rn[ib];    // straightforward difference
                if (rmx && mmx)
                {
                    // Compare intervals (see partial shuffle code in StcEvaluate.cpp)
                    int xn = __max(rn[ib], mn[ib]);     // max of mins
                    int nx = __min(rx[ib], mx[ib]);     // min of maxs
                    if (xn <= nx)
                        diff1 = 0;          // overlapping ranges -> no error
                    else
                        diff1 = (mn[ib] > rx[ib]) ?     // check sign
                        mn[ib] - rx[ib] :
                        rn[ib] - mx[ib];          // gap between intervals
                }
                int diff2 = (match_fn == eSD) ?    // squared or absolute difference
                    diff1 * diff1 : abs(diff1);
                diff_sum += diff2;
            }
            int diff3 = __min(diff_sum, cutoff);    // truncated difference
            __syncthreads();

            cost1[x] = diff3;        // store in temporary array
        }
    }
}

void MatchPixels(int w, int b, int interpolated,
    int* rmn, int* rmx,     // min/max of ref (ref if rmx == 0)
    int* mmn, int* mmx,     // min/max of mtc (mtc if mmx == 0)
    float* cost1,
    int disp,
    EStereoMatchFn match_fn,  // matching function
    int n,
    int s,
    int cutoff,
    int match_interval,
    int match_interpolated,
    int buffer_length,
    int cost1_length)
{
    dim3 gridSize, blockSize(BLOCKSIZE, 1, 1);
    gridSize.x = (unsigned int)ceil((float)(cost1_length) / (float)blockSize.x);

    float* cost1_d; // length n
    int* rmn_d;
    int* rmx_d;
    int* mmn_d;
    int* mmx_d;

    /*
    rmn = (match_interval) ? (match_interpolated) ? min0 : buf0 : buf0,
    rmx = (match_interval) ? (match_interpolated) ? max0 : buf0 : 0,
    mmn = (match_interval) ? min1 : buf1,
    mmx = (match_interval) ? max1 : 0,
    */

    // cost1 is the output and is not on device
    hipMalloc(&cost1_d, cost1_length*sizeof(float));

    // Input arrays host/device location is conditional
    if (match_interval)
    {
        if (match_interpolated)
        {
            // rmn = min0 and rmx = max0 already on device from Birchfield
            rmn_d = rmn;
            rmx_d = rmx;
        }
        else
        {
            // Birchfield with rmn = rmx = buf0 not run on device
            hipMalloc(&rmn_d, buffer_length*sizeof(int));
            hipMemcpy(rmn_d, rmn, buffer_length*sizeof(int), hipMemcpyHostToDevice);
            hipMalloc(&rmx_d, buffer_length*sizeof(int));
            hipMemcpy(rmx_d, rmx, buffer_length*sizeof(int), hipMemcpyHostToDevice);
        }
        // min1 and max1 already on device from Birchfield
        mmn_d = mmn;
        mmx_d = mmx;
    }
    else
    {
        // Birchfield not run, so buf0 and buf1 not on device
        hipMalloc(&rmn_d, buffer_length*sizeof(int));
        hipMemcpy(rmn_d, rmn, buffer_length*sizeof(int), hipMemcpyHostToDevice);
        rmx_d = rmx; // null
        hipMalloc(&mmn_d, buffer_length*sizeof(int));
        hipMemcpy(mmn_d, mmn, buffer_length*sizeof(int), hipMemcpyHostToDevice);
        mmx_d = mmx; // null
    }

    MatchPixelsKernel<<<gridSize, blockSize>>>(w, b, interpolated, rmn_d, rmx_d, mmn_d, mmx_d, cost1_d, disp, match_fn, n, s, cutoff, buffer_length, cost1_length);
    hipDeviceSynchronize();

    // Copy output back to host
    hipMemcpy(cost1, cost1_d, cost1_length*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(cost1_d);

    // Free device memory
    if (match_interval)
    {
        hipFree(rmn_d);
        hipFree(rmx_d);
        hipFree(mmn_d);
        hipFree(mmx_d);
    }
    else
    {
        // Birchfield not run, so buf0 and buf1 not on device
        hipFree(rmn_d);
        // rmx_d null
        hipFree(mmn_d);
        // mmx_d null
    }
}


__global__ void BoxFilterKernel(float* cost1, float* cost, int n, int w, int m_disp_n, int disp_den, int interpolated, int cost1_length, int cost_length)
{
    __shared__ float cost1_s[BLOCKSIZE];

    unsigned x = (threadIdx.x + blockIdx.x * blockDim.x) * disp_den;
    unsigned y = (threadIdx.y + blockIdx.y * blockDim.y) * m_disp_n;

    if (x < cost1_length)
        cost1_s[x] = cost1[x];
    __syncthreads();

    // Box filter if interpolated costs
    int dh = disp_den / 2;
    float box_scale = 1.0 / (2 * dh + 1);
    if (y < cost_length && x < cost1_length)
    {
        if (interpolated && disp_den > 1)
        {
            float sum = 0;
            for (int k = -dh; k <= dh; k++)
            {
                int l = __max(0, __min(n - 1, x + k));  // TODO: make more efficient
                sum += cost1_s[l];
            }
            cost[y] = int(box_scale * sum + 0.5);
        }
        else
            cost[y] = cost1_s[x];
    }
}

void BoxFilter(float* cost1, float* cost, int n, int w, int m_disp_n, int disp_den, int interpolated, int cost1_length, int cost_length)
{
    dim3 gridSize, blockSize(BLOCKSIZE, BLOCKSIZE, 1);
    gridSize.x = (unsigned int)ceil((float)(cost1_length) / (float)blockSize.x);
    gridSize.y = (unsigned int)ceil((float)(cost_length) / (float)blockSize.y);

    float* cost_d; // cost_length
    float* cost1_d; // cost1_length

    hipMalloc(&cost_d, cost_length*sizeof(float));
    hipMalloc(&cost1_d, cost1_length*sizeof(float));

    hipMemcpy(cost1_d, cost1, cost1_length*sizeof(float), hipMemcpyHostToDevice);

    BoxFilterKernel<<<gridSize, blockSize>>>(cost1_d, cost_d, n, w, m_disp_n, disp_den, interpolated, cost1_length, cost_length);
    hipDeviceSynchronize();

    hipMemcpy(cost, cost_d, cost_length*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(cost_d);
    hipFree(cost1_d);
}

// Parallelized Execution 2

__device__ int PixelCoordToAbs(ImageSizeStruct size, int x, int y, int band)
{
    return y * size.rowSize + x * size.pixSize + band * size.bandSize;
}

__global__ void LineProcessKernel(ImageStructUChar m_reference, ImageStructUChar m_matching, ImageStructFloat m_cost,
    int* buffer0, int* buffer1, int* min_bf0, int* max_bf0, int* min_bf1, int* max_bf1,
    int m_disp_den, int m_disp_n, int b, int w, int h, EStereoInterpFn match_interp, int match_interval, int match_interpolated, int n_interp,
    int m_frame_diff_sign, int disp_min, int m_disp_num, EStereoMatchFn match_fn, int match_max, float match_outside)
{
    unsigned y = (threadIdx.y + blockIdx.y * blockDim.y);
    unsigned buf_start = y * m_disp_den * b;

    // Process all of the lines
    if (y < h)
    {
        uchar* ref = &m_reference.image[PixelCoordToAbs(m_reference.imageSize, 0, y, 0)];
        uchar* mtc = &m_matching.image[PixelCoordToAbs(m_matching.imageSize, 0, y, 0)];
        int*  buf0 = &buffer0[buf_start];
        int*  buf1 = &buffer1[buf_start];
        int*  min0 = &min_bf0[buf_start];
        int*  max0 = &max_bf0[buf_start];
        int*  min1 = &min_bf1[buf_start];
        int*  max1 = &max_bf1[buf_start];

        // Fill the line buffers
        int x, l, m;
        for (x = 0, l = 0, m = 0; x < w; x++, m += m_disp_den*b)
        {
            for (int k = 0; k < b; k++, l++)
            {
                buf0[m + k] = ref[l];
                buf1[m + k] = mtc[l];
            }
        }

        // Interpolate the matching signal
        if (m_disp_den > 1)
        {
            InterpolateLine(buf1, m_disp_den, w, b, match_interp);
            InterpolateLine(buf0, m_disp_den, w, b, match_interp);
        }

        if (match_interval) {
            BirchfieldTomasiMinMax(buf1, min1, max1, n_interp, b);
            if (match_interpolated)
                BirchfieldTomasiMinMax(buf0, min0, max0, n_interp, b);
        }

        // Compute the costs, one disparity at a time
        for (int k = 0; k < m_disp_n; k++)
        {
            float* cost = &m_cost.image[PixelCoordToAbs(m_cost.imageSize, 0, y, k)];
            int disp = -m_frame_diff_sign * (m_disp_den * disp_min + k * m_disp_num);

            MatchLine(w, b, match_interpolated,
                (match_interval) ? (match_interpolated) ? min0 : buf0 : buf0,
                (match_interval) ? (match_interpolated) ? max0 : buf0 : 0,
                (match_interval) ? min1 : buf1,
                (match_interval) ? max1 : 0,
                cost,
                m_disp_n, disp, m_disp_den,
                match_fn,
                match_max,
                match_outside);
        }
    }
}

ImageSizeStruct PopulateImageSizeStruct(CImage image)
{
    ImageSizeStruct size;
    size.bands = image.Shape().nBands;
    size.height = image.Shape().height;
    size.width = image.Shape().width;
    size.bandSize = image.BandSize();
    size.pixSize = image.PixSize();
    size.rowSize = image.RowSize();

    return size;
}

void LineProcess(CByteImage m_reference, CByteImage m_matching, CFloatImage m_cost,
    int m_disp_den, int m_disp_n, int b, int w, int h, EStereoInterpFn match_interp, int* match_interval, int match_interpolated,
    int m_frame_diff_sign, int disp_min, int m_disp_num, EStereoMatchFn match_fn, int match_max, float* m_match_outside)
{
    // Allocate a buffer for interpolated values
    //  Note that we don't have to interpolate the ref image if we
    //  aren't using match_interpolated, but it's simpler to code this way.
    *match_interval = (match_interval ? 1 : 0);  // force to [0,1]
    int n_interp = m_disp_den * (w - 1) + 1;

    // Allocate working buffers
    iptr buffer0, buffer1, min_bf0, max_bf0, min_bf1, max_bf1;
    int buf_length = n_interp * b;
    int buf_size = h * buf_length * sizeof(int);
    
    hipMalloc(&buffer0, buf_size);
    hipMalloc(&buffer1, buf_size);
    hipMalloc(&min_bf0, buf_size);
    hipMalloc(&max_bf0, buf_size);
    hipMalloc(&min_bf1, buf_size);
    hipMalloc(&max_bf1, buf_size);

    // Allocate input and output image data
    uchar *m_ref_d, *m_match_d;
    float* m_cost_d;

    int m_ref_size = m_reference.ImageSize() * sizeof(uchar);
    int m_match_size = m_matching.ImageSize() * sizeof(uchar);
    int m_cost_size = m_cost.ImageSize() * sizeof(float);

    hipMalloc(&m_ref_d, m_ref_size);
    hipMalloc(&m_match_d, m_match_size);
    hipMalloc(&m_cost_d, m_cost_size);

    // Copy image data to device
    hipMemcpy(m_ref_d, &m_reference.Pixel(0, 0, 0), m_ref_size, hipMemcpyHostToDevice);
    hipMemcpy(m_match_d, &m_matching.Pixel(0, 0, 0), m_match_size, hipMemcpyHostToDevice);

    // Populate structs to hold picture info
    ImageStructUChar m_ref_struct, m_match_struct;
    ImageStructFloat m_cost_struct;

    m_ref_struct.imageSize = PopulateImageSizeStruct(m_reference);
    m_ref_struct.image = m_ref_d;
    m_match_struct.imageSize = PopulateImageSizeStruct(m_matching);
    m_match_struct.image = m_match_d;
    m_cost_struct.imageSize = PopulateImageSizeStruct(m_cost);
    m_cost_struct.image = m_cost_d;

    // Special value for border matches
    int worst_match = b * ((match_fn == eSD) ? 255 * 255 : 255);
    int cutoff = (match_fn == eSD) ? match_max * match_max : abs(match_max);
    *m_match_outside = __min(worst_match, cutoff);	// trim to cutoff

    dim3 gridSize, blockSize(1, BLOCKSIZE, 1);
    gridSize.y = (unsigned int)ceil((float)(h) / (float)blockSize.y);
    
    // Kernel call
    LineProcessKernel<<<gridSize, blockSize>>>(m_ref_struct, m_match_struct, m_cost_struct,
        buffer0, buffer1, min_bf0, max_bf0, min_bf1, max_bf1,
        m_disp_den, m_disp_n, b, w, h, match_interp, *match_interval, match_interpolated, n_interp,
        m_frame_diff_sign, disp_min, m_disp_num, match_fn, match_max, *m_match_outside);

    hipDeviceSynchronize();

    // Copy cost data to host
    hipMemcpy(&m_cost.Pixel(0, 0, 0), m_cost_d, m_cost_size, hipMemcpyDeviceToHost);

    // Free the memory
    hipFree(buffer0);
    hipFree(buffer1);
    hipFree(min_bf0);
    hipFree(max_bf0);
    hipFree(min_bf1);
    hipFree(max_bf1);

    hipFree(m_ref_d);
    hipFree(m_match_d);
    hipFree(m_cost_d);
}