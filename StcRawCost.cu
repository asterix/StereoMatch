#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "StereoMatcher.h"
#include "Warp1D.h"

#include <hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include ""

#include "StcRawCost.h"

extern Timer* profilingTimer2;

#define SHARED_MEM (0)

// Serial / Parallel implementation

__host__ __device__ void InterpolateLineCuda(int buf[], int s, int w, int nB, EStereoInterpFn match_interp)     // interpolation function
{
    // Interpolate the missing values
    float si = 1.0f / s;
    for (int x = 0; x < w - 1; x++)
    {
        for (int b = 0; b < nB; b++)
        {
            int* v = &buf[s*x*nB + b];
            float I0 = (float)v[0];
            float I1 = (float)v[s*nB];
            if (match_interp == eCubic) // cubic interpolation
            {
                float Im = (x > 0) ? v[-s*nB] :
                    (I0 - (I1 - I0));  // extend linearly
                float Ip = (x + 1 < w - 1) ? v[2 * s*nB] :
                    (I1 + (I1 - I0));  // extend linearly
                float sf = si;
                for (int is = 1; is < s; is++, sf += si)
                {
                    v += nB;
                    float Ii = CubicInterpolateCuda(sf, Im, I0, I1, Ip);
                    v[0] = int(Ii);
                }
            }
            else  // linear interpolation
            {
                float d = (I1 - I0) / (float)s;
                for (int is = 1; is < s; is++)
                {
                    v += nB;
                    I0 += d;
                    v[0] = int(I0);
                }
            }
        }
    }
}

__host__ __device__ float CubicInterpolateCuda(float x0, float v0, float v1, float v2, float v3)
{
    // See Szeliski & Ito, IEE Proc 133(6) 1986.
    float x1 = 1.0f - x0;
    float s0 = v2 - v0;     // slope matches central difference
    float s1 = v1 - v3;     // slope matches central difference
    float d1 = v2 - v1;
    float phi0 = d1 * (x0 * x0) * (2.0f * x1 + 1.0f);
    float phi1a = s0 * x0 * (x1 * x1);
    float phi1b = s1 * x1 * (x0 * x0);
    float v = v1 + phi0 + phi1a + phi1b;
    return v;
}

__host__ __device__ void BirchfieldTomasiMinMaxCuda(const int* buffer, int* min_buf, int* max_buf, const int w, const int b)
{
    // Compute for every (interpolated) pixel, the minimum and maximum
    //  values in the two half-intervals before and after it
    //  (see [Birchfield & Tomasi, PAMI 20(40), April 1998, p. 401]).

    // Process each band separaterly
    for (int k = 0; k < b; k++)
    {
        int Ir = buffer[k], b1 = buffer[k];
        for (int x = 0, l = k; x < w; x++, l += b)
        {
            int Il = Ir, b0 = b1;   // shift down previously computed values
            if (x < w - 1)
                b1 = buffer[l + b];
            Ir = (b0 + b1 + 1) / 2;   // interpolated half-value
            min_buf[l] = __min(Il, __min(b0, Ir));
            max_buf[l] = __max(Il, __max(b0, Ir));
        }
    }
}

__host__ __device__ void MatchLineCuda(MatchLineStruct args, float* cost, float* cost1)
{
    // Set up the starting addresses, pointers, and cutoff value
    int n = (args.w - 1)*args.m_disp_den + 1;             // number of reference pixels
    int s = (args.interpolated) ? 1 : args.m_disp_den;     // skip in reference pixels
    int cutoff = (args.match_fn == eSD) ? args.match_max * args.match_max : abs(args.match_max);
    // TODO:  cutoff is not adjusted for the number of bands...

    // Match valid pixels
    float  left_cost = BAD_COST;
    float right_cost = BAD_COST;

    for (int x = 0; x < n; x += s)
    {
        // Compute ref and match pointers
        cost1[x] = BAD_COST;
        int x_r = x, x_m = x + args.disp;
        if (x_m < 0 || x_m >= n)
            continue;
        int* rn = &(args.rmn[x_r*args.b]);    // pointer to ref or min pixel(s)
        int* rx = &(args.rmx[x_r*args.b]);    // pointer to ref    max pixel(s)
        int* mn = &(args.mmn[x_m*args.b]);    // pointer to mtc or min pixel(s)
        int* mx = &(args.mmx[x_m*args.b]);    // pointer to mtc    max pixel(s)
        int  diff_sum = 0;        // accumulated error

        // This code could be special-cased for b==1 for more efficiency...
        for (int ib = 0; ib < args.b; ib++)
        {
            int diff1 = mn[ib] - rn[ib];    // straightforward difference
            if (args.rmx && args.mmx)
            {
                // Compare intervals (see partial shuffle code in StcEvaluate.cpp)
                int xn = __max(rn[ib], mn[ib]);     // max of mins
                int nx = __min(rx[ib], mx[ib]);     // min of maxs
                if (xn <= nx)
                    diff1 = 0;          // overlapping ranges -> no error
                else
                    diff1 = (mn[ib] > rx[ib]) ?     // check sign
                    mn[ib] - rx[ib] :
                    rn[ib] - mx[ib];          // gap between intervals
            }
            int diff2 = (args.match_fn == eSD) ?    // squared or absolute difference
                diff1 * diff1 : abs(diff1);
            diff_sum += diff2;
        }
        int diff3 = __min(diff_sum, cutoff);    // truncated difference
        if (left_cost == BAD_COST)
            left_cost = (float)diff3;  // first cost computed
        right_cost = (float)diff3;     // last  cost computed
        cost1[x] = (float)diff3;        // store in temporary array
    }

    // Fill in the left and right edges
    if (UNDEFINED_COST)
        left_cost = right_cost = args.match_outside;
    for (int x = 0; x < n && cost1[x] == BAD_COST; x += s)
        cost1[x] = left_cost;
    for (int x = n - 1; x >= 0 && cost1[x] == BAD_COST; x -= s)
        cost1[x] = right_cost;

    // Box filter if interpolated costs
    int dh = args.m_disp_den / 2;
    float box_scale = 1.0 / (2 * dh + 1);
    for (int x = 0, y = 0; y < args.w*args.m_disp_n; x += args.m_disp_den, y += args.m_disp_n)
    {
        if (args.interpolated && args.m_disp_den > 1)
        {
            float sum = 0;
            for (int k = -dh; k <= dh; k++)
            {
                int l = __max(0, __min(n - 1, x + k));  // TODO: make more efficient
                sum += cost1[l];
            }
            cost[y] = (float)(int)(box_scale * sum + 0.5);
        }
        else
            cost[y] = cost1[x];
    }
}

// Kernel functions

__global__ void LineProcessKernel(ImageStructUChar m_reference, ImageStructUChar m_matching, ImageStructFloat m_cost,
    BufferStruct buffs, LineProcessStruct args)
{

#if SHARED_MEM
    extern __shared__ int shared_mem[];

    __shared__ int*  buf0_s;
    __shared__ int*  buf1_s;
    __shared__ int*  min0_s;
    __shared__ int*  max0_s;
    __shared__ int*  min1_s;
    __shared__ int*  max1_s;
    __shared__ float* cost1_s;
#endif

    unsigned y = (threadIdx.y + blockIdx.y * blockDim.y);
    int in_bounds = (y < args.h) ? 1 : 0;

    uchar* ref;
    uchar* mtc;

    int*  buf0;
    int*  buf1;
    int*  min0;
    int*  max0;
    int*  min1;
    int*  max1;
    float* cost1;

    // Process all of the lines
    if (in_bounds)
    {
#if SHARED_MEM
        // break apart shared memory
        buf0_s = shared_mem;
        buf1_s = buf0_s + buffs.buffer0.num_elems;
        min0_s = buf1_s + buffs.buffer1.num_elems;
        max0_s = min0_s + buffs.min_bf0.num_elems;
        min1_s = max0_s + buffs.max_bf0.num_elems;
        max1_s = min1_s + buffs.min_bf1.num_elems;
        cost1_s = (float*)(max1_s + buffs.max_bf1.num_elems);

        // reassign start based on row
        buf0 = &buf0_s[y * buffs.buffer0.width];
        buf1 = &buf1_s[y * buffs.buffer1.width];
        min0 = &min0_s[y * buffs.min_bf0.width];
        max0 = &max0_s[y * buffs.max_bf0.width];
        min1 = &min1_s[y * buffs.min_bf1.width];
        max1 = &max1_s[y * buffs.max_bf1.width];
        cost1 = &cost1_s[y * buffs.cost1.width];
#else
        buf0 = &(buffs.buffer0.array[y * buffs.buffer0.width]);
        buf1 = &(buffs.buffer1.array[y * buffs.buffer1.width]);
        min0 = &(buffs.min_bf0.array[y * buffs.min_bf0.width]);
        max0 = &(buffs.max_bf0.array[y * buffs.max_bf0.width]);
        min1 = &(buffs.min_bf1.array[y * buffs.min_bf1.width]);
        max1 = &(buffs.max_bf1.array[y * buffs.max_bf1.width]);
        cost1 = &(buffs.cost1.array[y * buffs.cost1.width]);
#endif

        ref = PixelAddress(m_reference, 0, y, 0);
        mtc = PixelAddress(m_matching, 0, y, 0);

        // Fill the line buffers
        for (int x = 0, l = 0, m = 0; x < args.w; x++, m += args.m_disp_den*args.b)
        {
            for (int k = 0; k < args.b; k++, l++)
            {
                buf0[m + k] = (int)ref[l];
                buf1[m + k] = (int)mtc[l];
            }
        }
    }
    __syncthreads();

    if (in_bounds)
    {
        // Interpolate the matching signal
        if (args.m_disp_den > 1)
        {
            InterpolateLineCuda(buf1, args.m_disp_den, args.w, args.b, args.match_interp);
            InterpolateLineCuda(buf0, args.m_disp_den, args.w, args.b, args.match_interp);
        }
    }
    __syncthreads();

    if (in_bounds)
    {
        if (args.match_interval) {
            BirchfieldTomasiMinMaxCuda(buf1, min1, max1, args.n_interp, args.b);
            if (args.match_interpolated)
                BirchfieldTomasiMinMaxCuda(buf0, min0, max0, args.n_interp, args.b);
        }
    }
    __syncthreads();

    if (in_bounds)
    {
        // Compute the costs, one disparity at a time
        for (int k = 0; k < args.m_disp_n; k++)
        {
            int disp = -args.m_frame_diff_sign * (args.m_disp_den * args.disp_min + k * args.m_disp_num);

            MatchLineStruct lineArgs = {
                args.w,
                args.b,
                args.match_interpolated, //match_interpolated
                (args.match_interval) ? (args.match_interpolated) ? min0 : buf0 : buf0, // rmn
                (args.match_interval) ? (args.match_interpolated) ? max0 : buf0 : 0, // rmx
                (args.match_interval) ? min1 : buf1, // mmn
                (args.match_interval) ? max1 : 0, // mmx
                args.m_disp_n,
                disp,
                args.m_disp_den,
                args.match_fn,
                args.match_max,
                args.match_outside
            };
            
            MatchLineCuda(lineArgs, PixelAddress(m_cost, 0, y, k), cost1);
        }
    }
}

void LineProcess(CByteImage m_reference, CByteImage m_matching, CFloatImage m_cost, LineProcessStruct args)
{
    
    profilingTimer2->startTimer();

    // Allocate working buffers
    BufferStruct buffs;
    int buf_width = args.n_interp * args.b; // size of one row (width)
    int buf_size = args.h * buf_width * sizeof(int); // in bytes

#if !SHARED_MEM    
    AllocateGPUMemory((void**)&(buffs.buffer0.array), buf_size, false);
    AllocateGPUMemory((void**)&(buffs.buffer1.array), buf_size, false);
    AllocateGPUMemory((void**)&(buffs.min_bf0.array), buf_size, false);
    AllocateGPUMemory((void**)&(buffs.max_bf0.array), buf_size, false);
    AllocateGPUMemory((void**)&(buffs.min_bf1.array), buf_size, false);
    AllocateGPUMemory((void**)&(buffs.max_bf1.array), buf_size, false);
#endif

    Populate2DArray(&buffs.buffer0, buf_width, args.h);
    Populate2DArray(&buffs.buffer1, buf_width, args.h);
    Populate2DArray(&buffs.min_bf0, buf_width, args.h);
    Populate2DArray(&buffs.max_bf0, buf_width, args.h);
    Populate2DArray(&buffs.min_bf1, buf_width, args.h);
    Populate2DArray(&buffs.max_bf1, buf_width, args.h);

    int cost1_width = ((args.w - 1)*args.m_disp_den + 1);
    int cost1_size = args.h * cost1_width * sizeof(float); // in bytes

#if !SHARED_MEM
    AllocateGPUMemory((void**)&(buffs.cost1.array), cost1_size, false);
#endif

    Populate2DArray(&buffs.cost1, cost1_width, args.h);

#if SHARED_MEM
    unsigned int total_buf_size = buf_size * 6 + cost1_size; // in bytes
#endif

    // Allocate input and output image data
    uchar* m_ref_d;
    uchar* m_match_d;
    float* m_cost_d;

    int m_ref_size = m_reference.ImageSize();
    int m_match_size = m_matching.ImageSize();
    int m_cost_size = m_cost.ImageSize();

    AllocateGPUMemory((void**)&m_ref_d, m_ref_size, false);
    AllocateGPUMemory((void**)&m_match_d, m_match_size, false);
    AllocateGPUMemory((void**)&m_cost_d, m_cost_size, false);
    

    // Copy image data to device
    CopyGPUMemory(m_ref_d, m_reference.PixelAddress(0, 0, 0), m_ref_size, true);
    CopyGPUMemory(m_match_d, m_matching.PixelAddress(0, 0, 0), m_match_size, true);

    // Populate structs to hold picture info
    ImageStructUChar m_ref_struct, m_match_struct;
    ImageStructFloat m_cost_struct;

    m_ref_struct.imageSize = PopulateImageSizeStruct(m_reference);
    m_ref_struct.image = m_ref_d;
    m_match_struct.imageSize = PopulateImageSizeStruct(m_matching);
    m_match_struct.image = m_match_d;
    m_cost_struct.imageSize = PopulateImageSizeStruct(m_cost);
    m_cost_struct.image = m_cost_d;

    printf("\nGPU Raw Costs memory allocation & copy: Time = %f ms\n", profilingTimer2->stopAndGetTimerValue());

    profilingTimer2->startTimer();

    // Block/Grid size
    dim3 gridSize, blockSize(1, BLOCKSIZE, 1);
    gridSize.y = (unsigned int)ceil((float)(args.h) / (float)blockSize.y);

    // Kernel call
#if !SHARED_MEM
    LineProcessKernel<<<gridSize, blockSize>>>(m_ref_struct, m_match_struct, m_cost_struct, buffs, args);
#else
    LineProcessKernel<<<gridSize, blockSize, total_buf_size>>>(m_ref_struct, m_match_struct, m_cost_struct, buffs, args);
#endif

    GPUERRORCHECK(hipDeviceSynchronize());

    printf("\nGPU Raw Costs kernel call: Time = %f ms\n", profilingTimer2->stopAndGetTimerValue());

    profilingTimer2->startTimer();

    // Copy cost data to host
    CopyGPUMemory(m_cost.PixelAddress(0, 0, 0), m_cost_d, m_cost_size, false);

    // Free the memory
#if !SHARED_MEM
    FreeGPUMemory(buffs.buffer0.array);
    FreeGPUMemory(buffs.buffer1.array);
    FreeGPUMemory(buffs.min_bf0.array);
    FreeGPUMemory(buffs.max_bf0.array);
    FreeGPUMemory(buffs.min_bf1.array);
    FreeGPUMemory(buffs.max_bf1.array);
    FreeGPUMemory(buffs.cost1.array);
#endif

    FreeGPUMemory(m_ref_d);
    FreeGPUMemory(m_match_d);
    FreeGPUMemory(m_cost_d);

    printf("\nGPU Raw Costs results transfer & clean-up: Time = %f ms\n", profilingTimer2->stopAndGetTimerValue());
}

// Helper functions

// Return the 1-D coordinate of the band pixel
__device__ int PixelCoordToAbs(ImageSizeStruct size, int x, int y, int band)
{
    return y * size.width * size.bands + x * size.bands + band;
}

// Return pointer to the address of the specified band pixel
__device__ uchar* PixelAddress(ImageStructUChar image, int x, int y, int band)
{
    return &image.image[PixelCoordToAbs(image.imageSize, x, y, band)];
}

// Return pointer to the address of the specified band pixel
__device__ float* PixelAddress(ImageStructFloat image, int x, int y, int band)
{
    return &image.image[PixelCoordToAbs(image.imageSize, x, y, band)];
}

// Populates the ImageSizeStruct from the provided CImage
ImageSizeStruct PopulateImageSizeStruct(CImage image)
{
    ImageSizeStruct size = {
        image.BandSize(),
        image.RowSize(),
        image.PixSize(),
        image.Shape().width,
        image.Shape().height,
        image.Shape().nBands
    };
    return size;
}

// Populates the TwoDIntArray structure
void Populate2DArray(TwoDIntArray* value, int width, int height)
{
    value->width = width;
    value->height = height;
    value->num_elems = width * height;
    value->size_bytes = width * height * sizeof(int);

}

// Populates the TwoDFloatArray structure
void Populate2DArray(TwoDFloatArray* value, int width, int height)
{
    value->width = width;
    value->height = height;
    value->num_elems = width * height;
    value->size_bytes = width * height * sizeof(float);
}