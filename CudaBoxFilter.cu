#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////////////////
// Implementation of convolution on GPU
// CudaBoxFilter.cu
// 
// 
// This file contains implementations of CUDA boxfiltering kernels and wrappers
//
//
//
//
// Created: 13-Dec-2014
//
///////////////////////////////////////////////////////////////////////////////////////////////////
#include "CudaBoxFilter.h"

// Profiling timers
extern Timer* profilingTimer2;
extern bool ZeroCopySupported;

                                                               
///////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel Prototypes
///////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ void BoxFilterXY(T* DevInBuffer, T* DevOutBuffer, CShape srcShape);



///////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel definitions
///////////////////////////////////////////////////////////////////////////////////////////////////

// BoxFiltering over variable window sizes
template <typename T>
__global__ void BoxFilterXY(T* DevInBuffer, T* DevOutBuffer, CShape srcShape)
{
    // Allow input tile to be stored in shared memory - Input is bigger than output tile
    __shared__ T Img[BOX_BLOCK_SIZE][BOX_BLOCK_SIZE];

    // Short notations
    int tix = threadIdx.x, tiy = threadIdx.y;
    float Sum;

    // Offset is basically the dimension of the extending halo
    int Offset = (int)BOX_WINDOW_SIZE / 2;

    int N_x = blockIdx.x * BOX_TILE_SIZE + tix - Offset;
    int N_y = blockIdx.y * BOX_TILE_SIZE + tiy - Offset;

    // Compute absolute indices just once for each thread
    int Abs_x = N_x * srcShape.nBands;
    int Abs_y = N_y * srcShape.nBands * srcShape.width;

    // Compute right bound
    int RBound = BOX_BLOCK_SIZE - Offset;

    // Subtract just once for each thread
    int OffX = tiy - Offset;
    int OffY = tix - Offset;

    // Boxfilter averaging elements
    int Avg = BOX_WINDOW_SIZE * BOX_WINDOW_SIZE;

    // Go over all depth layers (channels)
    for (int c = 0; c < srcShape.nBands; c++)
    {
        // If in bound of input N, then load the element, else load zero
        if ((N_x >= 0 && N_x < srcShape.width) && (N_y >= 0 && N_y < srcShape.height))
        {
            Img[tiy][tix] = DevInBuffer[Abs_y + Abs_x + c];
        }
        else
        {
            Img[tiy][tix] = 0.0f;
        }

        // Wait for all threads to load
        __syncthreads();

        // Clear sum
        Sum = 0.0f;

        // Accumulate and average - filtering non-halo threads in each block
        if ((tix >= Offset && tix < RBound) && (tiy >= Offset && tiy < RBound))
        {
            for (int i = 0; i < BOX_WINDOW_SIZE; i++)
            {
                for (int j = 0; j < BOX_WINDOW_SIZE; j++)
                {
                    Sum += Img[OffX + i][OffY + j];
                }
            }

            // Store the sum in P - Subset because N_x and N_y dest indices are applicable only to threads performing computation
            if ((N_x >= 0 && N_x < srcShape.width) && (N_y >= 0 && (N_y < srcShape.height)))
            {
                DevOutBuffer[Abs_y + Abs_x + c] = Sum/Avg;
            }
        }

        // Wait for all threads to finish computing
        __syncthreads();
    }
}



///////////////////////////////////////////////////////////////////////////////////////////////////
// CPU-GPU Host Wrappers
///////////////////////////////////////////////////////////////////////////////////////////////////
template <class T>
void CudaBoxFilterXY(CImageOf<T> src, CImageOf<T>& dst, int WindowSize)
{
    // Extract kernel details
    CShape srcShape = src.Shape();

    // GPU memories
    T *DevInBuffer, *DevOutBuffer;

    profilingTimer2->startTimer();
    // Get source and destination pointers
    int AllocSize;
    T *StartAddr = &src.Pixel(0, 0, 0);

    dst.ReAllocate(srcShape, false);
    T *DestStartAddr = &dst.Pixel(0, 0, 0);

    // Allocate memory to copy all image/cost-map channels to GPU
    if (!ZeroCopySupported)
    {
       AllocSize = sizeof(T) * srcShape.width * srcShape.height * srcShape.nBands;
       // Allocate in-data memory
       AllocateGPUMemory((void**)&DevInBuffer, AllocSize, false);
       // Allocate memory for output
       AllocateGPUMemory((void**)&DevOutBuffer, AllocSize, false);

       // Transfer everything to GPU
       CopyGPUMemory((void*)DevInBuffer, (void*)StartAddr, AllocSize, true);
    }
    else
    {
        // Zero-Copy - No Alloc - No Copy - No Free
        // Fast direct page-locked CPU-GPU access in unified physical memory architecture
        GPUERRORCHECK(hipHostGetDevicePointer((void **)&DevInBuffer, (void *)StartAddr, 0))
        GPUERRORCHECK(hipHostGetDevicePointer((void **)&DevOutBuffer, (void *)DestStartAddr, 0))
    }
    printf("\nMemCpy to GPU time = %f ms\n", profilingTimer2->stopAndGetTimerValue());

    // Set kernel parameters and launch kernel
    dim3 Grid(ceil((float)srcShape.width / BOX_TILE_SIZE), ceil((float)srcShape.height / BOX_TILE_SIZE), 1);
    dim3 Block(BOX_BLOCK_SIZE, BOX_BLOCK_SIZE, 1);

    profilingTimer2->startTimer();
    GPUERRORCHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoxFilterXY<float>), hipFuncCachePreferL1))
    BoxFilterXY<< <Grid, Block >> >(DevInBuffer, DevOutBuffer, srcShape);

    // Wait for all blocks to finish
    hipDeviceSynchronize();
    printf("\nBoxFilter kernel execution time = %f ms\n", profilingTimer2->stopAndGetTimerValue());

    // Copy computed elements back to CPU memory
    profilingTimer2->startTimer();
    // Only if traditional copy-back is needed
    if (!ZeroCopySupported)
    {
       CopyGPUMemory((void*)DestStartAddr, (void*)DevOutBuffer, AllocSize, false);
       
       // Free GPU memory
       FreeGPUMemory(DevInBuffer);
       FreeGPUMemory(DevOutBuffer);
    }
    printf("\nMemCpy from GPU time = %f ms\n", profilingTimer2->stopAndGetTimerValue());

}



///////////////////////////////////////////////////////////////////////////////////////////////////
// Templated functions' instantiation
///////////////////////////////////////////////////////////////////////////////////////////////////
template void CudaBoxFilterXY<float>(class CImageOf<float>, class CImageOf<float> &, int);
template void CudaBoxFilterXY<unsigned char>(class CImageOf<unsigned char>, class CImageOf<unsigned char> &, int);
template void CudaBoxFilterXY<int>(class CImageOf<int>, class CImageOf<int> &, int);

template __global__ void BoxFilterXY<float>(float *, float *, struct CShape);
